#include "hip/hip_runtime.h"
// Copyright (c) 2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "naive_histogram.h"

namespace naive_histogram {

using namespace ::dali;

__global__ void naive_histogram_kernel(
        const uint8_t *input, const int input_size, const float one_over_values_per_bin,
        int32_t *output) {
  for (int i = 0; i < input_size; i++) {
    output[static_cast<int>(input[i] * one_over_values_per_bin)]++;
  }
}


template<>
void NaiveHistogram<GPUBackend>::RunImpl(Workspace &ws) {
  const auto &input = ws.Input<GPUBackend>(0);
  const auto &shape = input.shape();
  auto &output = ws.Output<GPUBackend>(0);
  for (int sample_idx = 0; sample_idx < shape.num_samples(); sample_idx++) {  // Iterating over all samples in a batch.
    naive_histogram_kernel<<<1, 1, 0, ws.stream()>>>(
            input[sample_idx].data<uint8_t>(),
            volume(input.tensor_shape(sample_idx)),
            n_histogram_bins_ / 255.f,
            output[sample_idx].mutable_data<int32_t>()
    );
  }
}

}  // namespace naive_histogram